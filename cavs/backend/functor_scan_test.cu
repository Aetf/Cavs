#include "cavs/backend/functor_sort_scan.cuh"
#include "cavs/util/logging.h"
#include "cavs/util/macros_gpu.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <iostream>

using namespace backend;

const int SHARE_SIZE_LIMIT = 1 << 13;

int main() {
  //8k documents
  for (int Batch = 1; Batch <= 1 << 10; Batch <<= 1) {
    for (int N = 5; N <= 1 << 10; N <<= 1) {
      thrust::host_vector<float> h_vec(Batch*N, 2.f);
      thrust::device_vector<float> d_vec(h_vec);
      thrust::host_vector<float> h_vec_verify(h_vec);
      thrust::device_vector<float> d_vec_verify(h_vec);
      LOG(INFO) << "Testing with N = " << N
                << "\tand Batch = " << Batch << "\t...";
      int threadsPerBlock = N;
      int blocksPerGrid = Batch;
      CHECK(N <= SHARE_SIZE_LIMIT);
      CHECK(N == threadsPerBlock);//it is assumed in current implementation

      BatchedScan<SHARE_SIZE_LIMIT><<<blocksPerGrid, threadsPerBlock>>>(
          thrust::raw_pointer_cast(d_vec.data()),
          thrust::raw_pointer_cast(d_vec.data()),
          N);

      checkCudaError(hipDeviceSynchronize());
      checkCudaError(hipGetLastError());
      thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());

      for (int i = 0; i < Batch; i++) {
        thrust::inclusive_scan(
            d_vec_verify.begin()+i*N, d_vec_verify.begin()+(i+1)*N, 
            d_vec_verify.begin()+i*N);
      }
      thrust::copy(d_vec_verify.begin(), d_vec_verify.end(), h_vec_verify.begin());
      for (int i = 0; i < Batch; i++) {
        for (int j = 0; j < N; j++) {
          CHECK((h_vec[i*N+j] == h_vec_verify[i*N+j]))
               << "h_vec[" << i << "][" << j << "]: "
               << h_vec[i*N+j]
               << "\th_vec_verify[" << i << "][" << j << "]: "
               << h_vec_verify[i*N+j];
        }
      }
      LOG(INFO) << "Test Passed!";
    }
  }
}
