#include "cavs/midend/macros_gpu.h"
#include "cavs/backend/placeholder_op.h"

namespace cavs {

template <typename T> 
struct CUDAMemCopy {
  static void Compute(T* out, const T* in, size_t n) {
    checkCudaError(hipMemcpy(out, in, n*sizeof(T), hipMemcpyHostToDevice));
  }
};

REGISTER_OP_BUILDER(Key("Placeholder").Device("GPU"), PlaceholderOp<CUDAMemCopy<float>, float>);

} //namespace cavs

