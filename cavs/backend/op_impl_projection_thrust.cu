#include "hip/hip_runtime.h"
#include "cavs/backend/op_impl.h"
#include "cavs/midend/allocator.h"
#include "cavs/backend/cuda_common.h"
#include "cavs/midend/devices.h"
#include "cavs/proto/tensor_shape.pb.h"
#include "cavs/util/macros_gpu.h"

#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

namespace backend {

using ::midend::Allocator;
using ::midend::GetAllocator;
using ::midend::DeviceTypeToString;
using ::midend::Tensor;

template <typename T>
class ProjectionOpThrust: public OpImpl {
 public:
  explicit ProjectionOpThrust(const OpDef& def);
  ~ProjectionOpThrust();
  void Compute(OpContext* context) override;

 private:
  Allocator* alloc_;
  T* workspace;
  T* lamda;
};

template <typename T>
ProjectionOpThrust<T>::ProjectionOpThrust(const OpDef& def)
    : OpImpl(def), workspace(NULL), lamda(NULL) {
  alloc_ = GetAllocator(DeviceTypeToString(GPU));
  if (!lamda)
    lamda = alloc_->Allocate<T>(sizeof(T));
}

template <typename T>
ProjectionOpThrust<T>::~ProjectionOpThrust() {
  if (lamda)
    alloc_->Deallocate<char>((char*)lamda); 
}

template <typename T>
__global__ void FindMax(T *out, const T* mu, const T* mu_scan, int n) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    if ((mu[i] + (1.f - mu_scan[i])/(i+1)) > 0) {
      if (i == n-1 || mu[i+1] + (1.f - mu_scan[i+1]/(i+2) < 0)) 
        *out = 1.f/i*(1-mu_scan[i]);
    }
  }
}

template <typename T>
__global__ void GetOutput(T *x, const T* y, const T* lamda, int n) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    T tmp = y[i] + *lamda;
    if (tmp > 0)
      x[i] = tmp;
    else
      x[i] = 0;
  }
}

template <typename T>
void ProjectionOpThrust<T>::Compute(OpContext* context) {
  const Tensor& var_in = context->Input(0);
  Tensor* var_out = context->Output(0);
  /*if (!workspace)*/
    /*workspace = alloc_->Allocate<char>(var_in.count()*sizeof(T));*/
  /*checkCudaError(hipMemcpy(workspace,*/
        /*var_in.data<T>(), var_in.count()*sizeof(T),*/
        /*hipMemcpyDeviceToDevice));*/
  int n = var_in.count();
  thrust::device_ptr<T> dev_ptr(const_cast<T*>(var_in.data<T>()));
  thrust::device_vector<T> mu(dev_ptr, dev_ptr+n);
  thrust::sort(mu.begin(), mu.end());
  thrust::device_vector<T> mu_scan(n);
  thrust::inclusive_scan(mu.begin(), mu.end(), mu_scan.begin());
  /*T lamda;*/
  checkCudaError(hipDeviceSynchronize());
  FindMax<T><<<THREADS_PER_BLOCK, BLOCKS_PER_GRID(n)>>>(lamda,
      thrust::raw_pointer_cast(mu.data()), 
      thrust::raw_pointer_cast(mu_scan.data()),
      n);
  checkCudaError(hipDeviceSynchronize());
  GetOutput<T><<<THREADS_PER_BLOCK, BLOCKS_PER_GRID(n)>>>(
      var_out->mutable_data<T>(),
      var_in.data<T>(), 
      lamda,
      n);
  checkCudaError(hipDeviceSynchronize());
}

REGISTER_OP_IMPL_BUILDER(Key("Simplex").Device("GPU"), ProjectionOpThrust<float>);

} //namespace backend
