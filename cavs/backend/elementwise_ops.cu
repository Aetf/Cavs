#include "hip/hip_runtime.h"
#include "cavs/midend/macros_gpu.h"
#include "cavs/backend/elementwise_ops.h"

namespace cavs {

template <typename OP, typename T> 
__global__ void UnaryKernel(T* out, const T* inp, size_t n) {
  CUDA_1D_KERNEL_LOOP(i, n) { 
    out[i] = OP::Compute(inp[i]); 
  } 
}

template <typename OP, typename T> 
__global__ void BinaryKernel(T* out, const T* inp0, const T* inp1, size_t n) {
  CUDA_1D_KERNEL_LOOP(i, n) { 
    out[i] = OP::Compute(inp0[i], inp1[i]); 
  } 
}

template <typename OP, typename T>
struct CUDAUnaryFunctor {
  static void Compute(T* out, const T* inp, size_t n) {
    UnaryKernel<OP, T><<<THREADS_PER_BLOCK, BLOCKS_PER_GRID(n)>>>(
        out, inp, n);
  }
};

template <typename OP, typename T>
struct CUDABinaryFunctor {
  static void Compute(T* out, const T* inp0, const T* inp1, size_t n) {
    BinaryKernel<OP, T><<<THREADS_PER_BLOCK, BLOCKS_PER_GRID(n)>>>(
        out, inp0, inp1, n);
  }
};

#define CudaUnaryOpInstance(math, dtype)    \
    UnaryOp<CUDAUnaryFunctor<math<dtype>, dtype>, dtype> 
#define CudaBinaryOpInstance(math, dtype)   \
    BinaryOp<CUDABinaryFunctor<math<dtype>, dtype>, dtype> 

/*REGISTER_OP_BUILDER(Key("Add").Device("GPU"), UnaryOp<CUDAUnaryFunctor<math::Abs<float>, float>, float>);*/
REGISTER_OP_BUILDER(Key("Abs").Device("GPU"), CudaUnaryOpInstance(math::Abs, float));
REGISTER_OP_BUILDER(Key("Add").Device("GPU"), CudaBinaryOpInstance(math::Add, float));

} //namespace cavs
