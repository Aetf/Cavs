#include "hip/hip_runtime.h"
#include "cavs/backend/op_impl.h"
#include "cavs/backend/cuda_common.h"
#include "cavs/backend/cublas_wrapper.h"
/*#include "cavs/midend/devices.h"*/
#include "cavs/proto/tensor_shape.pb.h"
#include "cavs/util/macros_gpu.h"

namespace backend {

using ::midend::Tensor;

template <typename T>
class EmbeddingLookupOp: public OpImpl {
 public:
  explicit EmbeddingLookupOp(const OpDef& def) : OpImpl(def) {}
  void Compute(OpContext* context) override;
};

template <typename T>
__global__ void BatchedCopy(T *embedding,
    const T* data, const T* matrix,
    int embedding_size) {
  int output_offset = blockIdx.x*embedding_size;
  int matrix_offset = data[blockIdx.x]*embedding_size;
  for (int round = 0; round < (embedding_size+blockDim.x-1)/blockDim.x; round++) {
    int offset_within_vec = threadIdx.x + round*blockDim.x;
    if (offset_within_vec < embedding_size) {  
      embedding[output_offset+offset_within_vec] =
        matrix[matrix_offset+offset_within_vec];
    }
  }
}

template <typename T>
void EmbeddingLookupOp<T>::Compute(OpContext* context) {
  const Tensor& input = context->Input(0);
  const Tensor& embedding_matrix = context->Input(1);
  Tensor* embedding = context->Output(0);

  CHECK(embedding_matrix.dims() == 2);
  int vocabulary_size = embedding_matrix.dims(0);
  int embedding_size  = embedding_matrix.dims(1);
  CHECK(vocabulary_size >= embedding_size);
  CHECK(embedding->dims() == input.dims()+1);
  for (int i = 0; i < input.dims(); i++)
    CHECK(embedding->dims(i) == input.dims(i));
  CHECK(embedding->dims(embedding->dims()-1) == embedding_size);

  int slices = input.count();
  const int MAX_THREADS_IN_BLOCK = 1 << 10;
  int threadsPerBlock = (MAX_THREADS_IN_BLOCK > embedding_size) ?
                         embedding_size : MAX_THREADS_IN_BLOCK;
  int blocksPerGrid = slices;
  BatchedCopy<<<blocksPerGrid, threadsPerBlock>>>(
      embedding->mutable_data<T>(),
      input.data<T>(), embedding_matrix.data<T>(),
      embedding_size);

  input.DebugNumerical<T>();
  embedding_matrix.DebugNumerical<T>();
  embedding->DebugNumerical<T>();
}

template <typename T>
class EmbeddingLookupGradOp: public OpImpl {
 public:
  explicit EmbeddingLookupGradOp(const OpDef& def) : OpImpl(def) {}
  void Compute(OpContext* context) override;
};

template <typename T>
__global__ void BatchedSparseUpdate(T *dMatrix,
    const T* data, const T* dY,
    int embedding_size) {
  int dY_offset = blockIdx.x*embedding_size;
  int dMatrix_offset = data[blockIdx.x]*embedding_size;
  for (int round = 0; round < (embedding_size+blockDim.x-1)/blockDim.x; round++) {
    int offset_within_vec = threadIdx.x + round*blockDim.x;
    if (offset_within_vec < embedding_size) {  
      atomicAdd(&(dMatrix[dMatrix_offset+offset_within_vec]), 
        dY[dY_offset+offset_within_vec]);
    }
  }
}

template <typename T>
void EmbeddingLookupGradOp<T>::Compute(OpContext* context) {
  const Tensor& dY = context->Input(0);
  const Tensor& input = context->Input(1);
  Tensor* dMatrix= context->Output(0);
  //we don't calculate the dX, because dX is not passed backward

  CHECK(dMatrix->dims() == 2);
  int vocabulary_size = dMatrix->dims(0);
  int embedding_size  = dMatrix->dims(1);
  CHECK(vocabulary_size >= embedding_size);
  CHECK(dY.dims() == input.dims()+1);
  for (int i = 0; i < input.dims(); i++)
    CHECK(dY.dims(i) == input.dims(i));
  CHECK(dY.dims(dY.dims()-1) == embedding_size);

  checkCudaError(hipMemset(dMatrix->mutable_data<T>(), 0, 
                            dMatrix->count()*sizeof(T)));
  int slices = input.count();
  const int MAX_THREADS_IN_BLOCK = 1 << 10;
  int threadsPerBlock = (MAX_THREADS_IN_BLOCK > embedding_size) ?
                         embedding_size : MAX_THREADS_IN_BLOCK;
  int blocksPerGrid = slices;
  BatchedSparseUpdate<<<blocksPerGrid, threadsPerBlock>>>(
      dMatrix->mutable_data<T>(),
      input.data<T>(), dY.data<T>(),
      embedding_size);

  //A.DebugNumerical<T>();
  //B.DebugNumerical<T>();
  //C->DebugNumerical<T>();
}

REGISTER_OP_IMPL_BUILDER(Key("EmbeddingLookup").Device("GPU"), EmbeddingLookupOp<float>);
REGISTER_OP_IMPL_BUILDER(Key(GetGradientName("EmbeddingLookup")).Device("GPU"), EmbeddingLookupGradOp<float>);

} //namespace backend

