#include "cavs/backend/op_impl.h"
#include "cavs/backend/cuda_common.h"
#include "cavs/midend/allocator.h"
#include "cavs/midend/devices.h"
#include "cavs/proto/tensor_shape.pb.h"
#include "cavs/util/macros_gpu.h"
#include "cavs/util/cudnn_types.h"

namespace backend {
  
using ::midend::Allocator;
using ::midend::GetAllocator;
using ::midend::DeviceTypeToString;
using ::midend::Tensor;

template <typename T>
class PoolingOpCudnn : public OpImpl {
 public:
  explicit PoolingOpCudnn(const OpDef& def);
  void Compute(OpContext* context) override;

 private:
  hipdnnTensorDescriptor_t x_desc_, y_desc_;
  hipdnnPoolingDescriptor_t pooling_desc_;
  /*hipdnnPoolingMode_t mode_;*/
  int k_;
  int stride_;
};

template <typename T>
PoolingOpCudnn<T>::PoolingOpCudnn(const OpDef& def)
  : OpImpl(def) {
  k_ = GetSingleArg<int>("k");
  stride_ = GetSingleArg<int>("stride");
  checkCUDNNError(hipdnnCreateTensorDescriptor(&x_desc_));
  checkCUDNNError(hipdnnCreateTensorDescriptor(&y_desc_));
  checkCUDNNError(hipdnnCreatePoolingDescriptor(&pooling_desc_));
  checkCUDNNError(hipdnnSetPooling2dDescriptor(pooling_desc_,
      HIPDNN_POOLING_MAX, HIPDNN_NOT_PROPAGATE_NAN,
      k_, k_, 0, 0, stride_, stride_));
}

template <typename T>
void PoolingOpCudnn<T>::Compute(OpContext* context) {
  const Tensor& x = context->Input(0);
  Tensor* y = context->Output(0);
  int XN = x.dims(0);
  int XC = x.dims(1);
  int XH = x.dims(2);
  int XW = x.dims(3);
  int YN = y->dims(0);
  int YC = y->dims(1);
  int YH = y->dims(2);
  int YW = y->dims(3);
  CHECK(YN == XN);
  CHECK(YC == XC);
  checkCUDNNError(hipdnnSetTensor4dDescriptor(
      x_desc_,
      HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
      XN, XC, XH, XW));
  checkCUDNNError(hipdnnSetTensor4dDescriptor(
      y_desc_,
      HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
      YN, YC, YH, YW));
  float alpha = 1.f, beta = 0.f;
  checkCUDNNError(hipdnnPoolingForward(
      CudaCommon::cudnnHandle(), pooling_desc_,
      &alpha, x_desc_, x.data<T>(),
      &beta, y_desc_, y->mutable_data<T>()));
}

template <typename T>
class PoolingOpCudnnGrad : public OpImpl {
 public:
  explicit PoolingOpCudnnGrad(const OpDef& def);
  void Compute(OpContext* context) override;

 private:
  hipdnnTensorDescriptor_t x_desc_, y_desc_;
  hipdnnPoolingDescriptor_t pooling_desc_;
  /*hipdnnPoolingMode_t mode_;*/
  int k_;
  int stride_;
};

template <typename T>
PoolingOpCudnnGrad<T>::PoolingOpCudnnGrad(const OpDef& def)
  : OpImpl(def) {
  k_ = GetSingleArg<int>("k");
  stride_ = GetSingleArg<int>("stride");
  checkCUDNNError(hipdnnCreateTensorDescriptor(&x_desc_));
  checkCUDNNError(hipdnnCreateTensorDescriptor(&y_desc_));
  checkCUDNNError(hipdnnCreatePoolingDescriptor(&pooling_desc_));
  checkCUDNNError(hipdnnSetPooling2dDescriptor(pooling_desc_,
      HIPDNN_POOLING_MAX, HIPDNN_NOT_PROPAGATE_NAN,
      k_, k_, 0, 0, stride_, stride_));
}

template <typename T>
void PoolingOpCudnnGrad<T>::Compute(OpContext* context) {
  const Tensor& y = context->Input(0);
  const Tensor& dy = context->Input(1);
  const Tensor& x = context->Input(2);
  Tensor* dx = context->Output(0);
  int YN = y.dims(0);
  int YC = y.dims(1);
  int YH = y.dims(2);
  int YW = y.dims(3);
  int XN = x.dims(0);
  int XC = x.dims(1);
  int XH = x.dims(2);
  int XW = x.dims(3);
  CHECK(YN == XN);
  CHECK(YC == XC);
  checkCUDNNError(hipdnnSetTensor4dDescriptor(
      x_desc_,
      HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
      XN, XC, XH, XW));
  checkCUDNNError(hipdnnSetTensor4dDescriptor(
      y_desc_,
      HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
      YN, YC, YH, YW));
  float alpha = 1.f, beta = 0.f;
  checkCUDNNError(hipdnnPoolingBackward(
      CudaCommon::cudnnHandle(), pooling_desc_,
      &alpha, y_desc_, y.data<T>(),
      y_desc_, dy.data<T>(),
      x_desc_, x.data<T>(),
      &beta,
      x_desc_, dx->mutable_data<T>()));
}

} //namespace backend

