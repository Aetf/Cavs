#include "hip/hip_runtime.h"
#include "cavs/backend/functor_elementwise.h"
#include "cavs/backend/cuda_common.h"
#include "cavs/backend/op_impl.h"
#include "cavs/midend/tensor.h"
#include "cavs/midend/op_context.h"
#include "cavs/proto/tensor_shape.pb.h"
#include "cavs/util/macros_gpu.h"

namespace backend {

using ::midend::OpContext;
using ::midend::Tensor;

template <typename T> 
__global__ void SGDKernel(T* out, const T* inp0, const T* inp1,
    const float lr,size_t n) {
  CUDA_1D_KERNEL_LOOP(i, n) { 
    out[i] = inp0[i] - lr*inp1[i]; 
  } 
}

template <typename T>
class SGDOpImpl : public OpImpl {
 public:
  explicit SGDOpImpl(const OpDef& def)
    : OpImpl(def), lr_(0.f) {
      lr_ = GetSingleArg<float>(def, "learning_rate");
      /*CHECK(lr > 0);*/
      LOG(INFO) << "learning_rate = " << lr_;
  }

  void Compute(OpContext* context) override {
    const Tensor& inp0 = context->Input(0);
    const Tensor& inp1 = context->Input(1);
    /*inp0.DebugNumerical<T>();*/
    /*LOG(INFO) << "\n\n";*/
    /*inp1.DebugNumerical<T>();*/
    /*LOG(INFO) << "\n\n";*/
    Tensor* out = context->Output(0);
    int n = out->count();
    SGDKernel<T><<<BLOCKS_PER_GRID(n), THREADS_PER_BLOCK>>> (
        out->mutable_data<T>(),
        inp0.data<T>(), inp1.data<T>(), lr_, n);
    /*out->DebugNumerical<T>();*/
    /*LOG(INFO) << "\n\n";*/
  }

 private:
  float lr_;
};

REGISTER_OP_IMPL_BUILDER(Key("SGD").Device("GPU"), SGDOpImpl<float>);

} //namespace backend
