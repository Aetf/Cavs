#include "cavs/backend/op_impl_variable.h"
#include "cavs/util/macros_gpu.h"

namespace backend {

template <typename T> 
struct CUDAZeroFiller {
  static void Compute(T* out, size_t n) {
    checkCudaError(hipMemset(out, 0, n*sizeof(T)));
  }
};

REGISTER_OP_IMPL_BUILDER(Key("Variable").Device("GPU"), VariableOpImpl<CUDAZeroFiller<float>, float>);

} //namespace backend
