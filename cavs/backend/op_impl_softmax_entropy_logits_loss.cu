#include "hip/hip_runtime.h"
#include "cavs/backend/op_impl.h"
#include "cavs/backend/cuda_common.h"
#include "cavs/backend/cublas_wrapper.h"
#include "cavs/midend/allocator.h"
#include "cavs/midend/devices.h"
#include "cavs/proto/tensor_shape.pb.h"
#include "cavs/util/macros_gpu.h"
#include "cavs/util/cudnn_types.h"

namespace backend {

using ::midend::Allocator;
using ::midend::GetAllocator;
using ::midend::DeviceTypeToString;
using ::midend::Tensor;

class SoftmaxEntropyLogitsOpCudnnBase : public OpImpl {
 public:
  explicit SoftmaxEntropyLogitsOpCudnnBase(const OpDef& def);
  ~SoftmaxEntropyLogitsOpCudnnBase();

 protected:
  hipdnnTensorDescriptor_t x_desc_, y_desc_;
  hipdnnTensorDescriptor_t label_desc_;
};

SoftmaxEntropyLogitsOpCudnnBase::SoftmaxEntropyLogitsOpCudnnBase(const OpDef& def)
    : OpImpl(def) {
  checkCUDNNError(hipdnnCreateTensorDescriptor(&x_desc_));    
  checkCUDNNError(hipdnnCreateTensorDescriptor(&y_desc_));    
  checkCUDNNError(hipdnnCreateTensorDescriptor(&label_desc_));    
}

SoftmaxEntropyLogitsOpCudnnBase::~SoftmaxEntropyLogitsOpCudnnBase() {
  checkCUDNNError(hipdnnDestroyTensorDescriptor(x_desc_));
  checkCUDNNError(hipdnnDestroyTensorDescriptor(y_desc_));
  checkCUDNNError(hipdnnDestroyTensorDescriptor(label_desc_));
}

template <typename T>
class SoftmaxEntropyLogitsOpCudnn : public SoftmaxEntropyLogitsOpCudnnBase {
 public:
  explicit SoftmaxEntropyLogitsOpCudnn(const OpDef& def) 
      : SoftmaxEntropyLogitsOpCudnnBase(def) {}
  void Compute(OpContext* context) override;
};

template <typename T>
void SoftmaxEntropyLogitsOpCudnn<T>::Compute(OpContext* context) {
  const Tensor& x = context->Input(0);
  Tensor* y = context->Output(0);
  CHECK(x.dims() == y->dims());
  CHECK(x.dims() == 2);
  CHECK(x.dims(0) == y->dims(0));
  CHECK(x.dims(1) == y->dims(1));

  int XN, YN, XC, YC, XH, YH, XW, YW;
  XN = YN = x.dims(0);
  XC = YC = x.dims(1);
  XH = YH = 1;
  XW = YW = 1;

  float alpha = 1.f, beta = 0.f;
  checkCUDNNError(hipdnnSetTensor4dDescriptor(x_desc_,
                  HIPDNN_TENSOR_NCHW, DataTypeToCudnnType<T>::value,
                  XN, XC, XH, XW));
  checkCUDNNError(hipdnnSetTensor4dDescriptor(y_desc_,
                  HIPDNN_TENSOR_NCHW, DataTypeToCudnnType<T>::value,
                  YN, YC, YH, YW));
  checkCUDNNError(hipdnnSoftmaxForward(CudaCommon::cudnnHandle(),
                  HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL,
                  &alpha, x_desc_, x.data<T>(),
                  &beta, y_desc_, y->mutable_data<T>()));
  x.DebugNumerical<T>();
  y->DebugNumerical<T>();
}

template <typename T>
class SoftmaxEntropyLossOpCudnn : public SoftmaxEntropyLogitsOpCudnnBase {
 public:
  explicit SoftmaxEntropyLossOpCudnn(const OpDef& def) 
      : SoftmaxEntropyLogitsOpCudnnBase(def),
        workspace_(NULL), workspace_size_(0) {
    alloc_ = GetAllocator(DeviceTypeToString(GPU));
  }
  void Compute(OpContext* context) override;

 private:
  T* workspace_;
  size_t workspace_size_;
  Allocator* alloc_;
};

template <typename T>
__global__ void SoftmaxEntropyLossForwardKernel(
    T* output, const T* softmax, const T* label,
    int label_elements, int prediction_range) {
  //output and softmax can be the same buffer(in-place)
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= label_elements)  return;
  const int label_value = static_cast<int>(label[idx]);
  output[idx] = -log(softmax[idx*prediction_range+label_value]);
}

template <typename T>
void SoftmaxEntropyLossOpCudnn<T>::Compute(OpContext* context) {
  const Tensor& x = context->Input(0);
  const Tensor& label = context->Input(1);
  Tensor* y = context->Output(1);

  CHECK(x.dims() == 2 && label.dims() == 2 && y->dims() == 2);
  CHECK(x.dims(0) == label.dims(0) && label.dims(0) == y->dims(0));
  CHECK(label.dims(1) == y->dims(1) && y->dims(1) == 1);

  int XN = x.dims(0);
  int XC = x.dims(1);
  int XH = 1;
  int XW = 1;

  if (workspace_size_ != XN*XC*sizeof(T)) {
    workspace_size_ = XN*XC*sizeof(T);
    if (workspace_) 
        alloc_->Deallocate<char>((char*)workspace_); 
    workspace_ = (T*)alloc_->Allocate<char>(workspace_size_);
    checkCUDNNError(hipdnnSetTensor4dDescriptor(x_desc_,
                    HIPDNN_TENSOR_NCHW, DataTypeToCudnnType<T>::value,
                    XN, XC, XH, XW));
  }
  float alpha = 1.f, beta = 0.f;
  checkCUDNNError(hipdnnSoftmaxForward(CudaCommon::cudnnHandle(),
                  HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL,
                  &alpha, x_desc_, x.data<T>(),
                  &beta, x_desc_, workspace_));
  int n = label.count();
  SoftmaxEntropyLossForwardKernel<<<BLOCKS_PER_GRID(n), THREADS_PER_BLOCK>>>(
        y->mutable_data<T>(), workspace_, label.data<T>(), n, XC);

  x.DebugNumerical<T>();
  y->DebugNumerical<T>();
}

template <typename T>
class SoftmaxEntropyLogitsOpCudnnGrad : public SoftmaxEntropyLogitsOpCudnnBase {
 public:
  explicit SoftmaxEntropyLogitsOpCudnnGrad(const OpDef& def) 
      : SoftmaxEntropyLogitsOpCudnnBase(def) {}
  void Compute(OpContext* context) override;
};

template <typename T>
__global__ void SoftmaxEntropyLogitsBackwardKernel(T* dx, 
    const T* y, const T* label,
    int logits_elements, int prediction_range) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= logits_elements)  return;
  const int label_value = static_cast<int>(label[idx/prediction_range]);
  //Through the formula of cross-entropy,
  //the derivation of dx can be denoted as follows:
  //(I deduce it for a whole noon!)
  if (label_value == idx%prediction_range)
    dx[idx] = y[idx] - 1;
  else
    dx[idx] = y[idx];
}

template <typename T>
void SoftmaxEntropyLogitsOpCudnnGrad<T>::Compute(OpContext* context) {
  const Tensor& y = context->Input(0);
  const Tensor& label = context->Input(1);
  Tensor* dx = context->Output(0);

  CHECK(dx->dims() == y.dims());
  CHECK(dx->dims() == label.dims());
  CHECK(dx->dims() == 2);

  int NY = y.dims(0);
  int CY = y.dims(1);
  int NLabel = label.dims(0);
  int CLabel = label.dims(1);
  int NX = dx->dims(0);
  int CX = dx->dims(1);
  CHECK(CLabel == 1);
  CHECK(NX == NY);
  CHECK(CX == CY);
  int n = y.count();
  SoftmaxEntropyLogitsBackwardKernel<T><<<BLOCKS_PER_GRID(n), THREADS_PER_BLOCK>>>(
        dx->mutable_data<T>(), y.data<T>(), label.data<T>(), n, CY);
  T scale_gradient = 1.f/NX;
  ScalCublasWrapper(dx->count(), &scale_gradient, dx->mutable_data<T>());
  checkCudaError(hipDeviceSynchronize());
  y.DebugNumerical<T>();
  label.DebugNumerical<T>();
  dx->DebugNumerical<T>();
}

template <typename T>
class SoftmaxEntropyLossOpCudnnGrad : public SoftmaxEntropyLogitsOpCudnnBase {
 public:
  explicit SoftmaxEntropyLossOpCudnnGrad(const OpDef& def) 
      : SoftmaxEntropyLogitsOpCudnnBase(def),
        workspace_(NULL), workspace_size_(0) {
    alloc_ = GetAllocator(DeviceTypeToString(GPU));
  }
  void Compute(OpContext* context) override;

 private:
  T* workspace_;
  size_t workspace_size_;
  Allocator* alloc_;
};

template <typename T>
void SoftmaxEntropyLossOpCudnnGrad<T>::Compute(OpContext* context) {
  const Tensor& x = context->Input(0);
  const Tensor& label = context->Input(1);
  Tensor* dx = context->Output(0);

  CHECK(dx->dims() == 2 && label.dims() == 2 && x.dims() == 2);
  CHECK(x.dims(0) == label.dims(0) && label.dims(0) == dx->dims(0));
  CHECK(x.dims(1) == dx->dims(1) && label.dims(1) == 1);

  int XN = x.dims(0);
  int XC = x.dims(1);
  int XH = 1;
  int XW = 1;

  if (workspace_size_ != XN*XC*sizeof(T)) {
    workspace_size_ = XN*XC*sizeof(T);
    if (workspace_) 
        alloc_->Deallocate<char>((char*)workspace_); 
    workspace_ = (T*)alloc_->Allocate<char>(workspace_size_);
    checkCUDNNError(hipdnnSetTensor4dDescriptor(x_desc_,
                    HIPDNN_TENSOR_NCHW, DataTypeToCudnnType<T>::value,
                    XN, XC, XH, XW));
  }
  float alpha = 1.f, beta = 0.f;
  checkCUDNNError(hipdnnSoftmaxForward(CudaCommon::cudnnHandle(),
                  HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL,
                  &alpha, x_desc_, x.data<T>(),
                  &beta, x_desc_, workspace_));
  int n = x.count();
  SoftmaxEntropyLogitsBackwardKernel<T><<<BLOCKS_PER_GRID(n), THREADS_PER_BLOCK>>>(
        dx->mutable_data<T>(), workspace_, label.data<T>(), n, XC);
  T scale_gradient = 1.f/XN;
  ScalCublasWrapper(dx->count(), &scale_gradient, dx->mutable_data<T>());
  checkCudaError(hipDeviceSynchronize());

  label.DebugNumerical<T>();
  dx->DebugNumerical<T>();
}

REGISTER_OP_IMPL_BUILDER(Key("SoftmaxEntropyLogits").Device("GPU"),
    SoftmaxEntropyLogitsOpCudnn<float>);
REGISTER_OP_IMPL_BUILDER(Key(GetGradientName("SoftmaxEntropyLogits")).Device("GPU"),
    SoftmaxEntropyLogitsOpCudnnGrad<float>);

REGISTER_OP_IMPL_BUILDER(Key("SoftmaxEntropyLoss").Device("GPU"),
    SoftmaxEntropyLossOpCudnn<float>);
REGISTER_OP_IMPL_BUILDER(Key(GetGradientName("SoftmaxEntropyLoss")).Device("GPU"),
    SoftmaxEntropyLossOpCudnnGrad<float>);

} //namespace backend

