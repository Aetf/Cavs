#include "cavs/backend/op_impl.h"
#include "cavs/midend/tensor.h"
#include "cavs/util/macros_gpu.h"

using std::vector;
using std::string;
using ::midend::Tensor;

namespace backend {

template <typename T>
class SliceOpImpl : public OpImpl {
 public:
  explicit SliceOpImpl(const OpDef& def) :
    OpImpl(def), split_(-1), index_(-1), offset_(-1), stride_(-1) {
    CHECK(!GetSingleArg<bool>(op_def_, "ShareMemory", false));
    if (GetSingleArg(def, "Split", 0) != 0) {
      split_ = GetSingleArg<int>(def, "Split"); 
      index_ = GetSingleArg<int>(def, "Index"); 
      CHECK(split_ > 0);
      CHECK(index_ >= 0);
    }else {
      offset_ = GetSingleArg<int>(def, "Offset");
      stride_ = GetSingleArg<int>(def, "Stride");
      CHECK(offset_ >= 0);
      CHECK(stride_ > 0);
    }
  }
  void Compute(OpContext* context) override;

 private:
  int offset_;
  int stride_;
  int split_;
  int index_;
};

template <typename T>
void SliceOpImpl<T>::Compute(OpContext* context) {
  const Tensor& x = context->Input(0);
  Tensor* y = context->Output(0);

  if (offset_ < 0) {
    CHECK(x.count()% split_ == 0) << x.count() << "\t" << split_;
    stride_ = x.count() / split_;
    offset_ = x.count() / split_ * index_;
  }
  CHECK(stride_ == y->count());

  checkCudaError(hipMemcpy(y->mutable_data<T>(),
                            x.data<T>()+offset_,
                            stride_*sizeof(T),
                            hipMemcpyDeviceToDevice));
}

template <typename T>
class ConcatOpImpl : public OpImpl {
 public:
  explicit ConcatOpImpl(const OpDef& def) : OpImpl(def) {}
  void Compute(OpContext* context) override {
    Tensor* out = context->Output(0);
    CHECK(out->count() > 0);
    int copied_count = 0;
    for (int i = 0; i < context->InputSize(); i++) {
      const Tensor& inp = context->Input(i);
      CHECK(inp.count() > 0);
      CHECK(copied_count + inp.count() <= out->count());
      checkCudaError(hipMemcpy(out->mutable_data<T>()+copied_count,
                                inp.data<T>(),
                                inp.count()*sizeof(T),
                                hipMemcpyDeviceToDevice));
      copied_count += inp.count();
      inp.DebugNumerical<T>();
    } 
    CHECK(out->count() == copied_count);
    out->DebugNumerical<T>();
  }
};

template <typename T>
class SliceAllOpImpl : public OpImpl {
 public:
  explicit SliceAllOpImpl(const OpDef& def) : OpImpl(def) {}
  void Compute(OpContext* context) override {
    CHECK(context->InputSize() == context->OutputSize()+1);
    const Tensor& input = context->Input(0);
    input.DebugNumerical<T>();
    CHECK(input.count() > 0);
    int copied_count = 0;
    for (int i = 0; i < context->OutputSize(); i++) {
      const Tensor& inp_check = context->Input(i+1);
      Tensor* out = context->Output(i);
      CHECK(inp_check.count() == out->count());
      CHECK(copied_count + out->count() <= input.count());
      checkCudaError(hipMemcpy(out->mutable_data<T>(),
                                input.data<T>()+copied_count,
                                out->count()*sizeof(T),
                                hipMemcpyDeviceToDevice));
      copied_count += out->count();
      out->DebugNumerical<T>();
    } 
    CHECK(input.count() == copied_count);
  }
};

REGISTER_OP_IMPL_BUILDER(Key("Slice").Device("GPU"),    SliceOpImpl<float>);
REGISTER_OP_IMPL_BUILDER(Key("Concat").Device("GPU"),   ConcatOpImpl<float>);
REGISTER_OP_IMPL_BUILDER(Key("SliceAll").Device("GPU"), SliceAllOpImpl<float>);

} //namespace backend
