#include "cavs/backend/op_impl.h"
#include "cavs/backend/cuda_common.h"
#include "cavs/midend/allocator.h"
#include "cavs/midend/devices.h"
#include "cavs/proto/tensor_shape.pb.h"
#include "cavs/util/macros_gpu.h"
#include "cavs/util/cudnn_types.h"

#include <string>
#include <vector>

using std::string;
using std::vector;

namespace backend {

using ::midend::Allocator;
using ::midend::GetAllocator;
using ::midend::DeviceTypeToString;
using ::midend::Tensor;

template <typename T>
class RNNOpCudnnBase : public OpImpl {
 public:
  explicit RNNOpCudnnBase(const OpDef& def);
  ~RNNOpCudnnBase(); 

  virtual void InitCUDNN(int seq_length, int batch,
      int input_size, int rnn_params_count);

 protected:
  vector<hipdnnTensorDescriptor_t> x_desc_ , y_desc_ ;
  hipdnnTensorDescriptor_t  hx_desc_, hy_desc_;
  hipdnnTensorDescriptor_t  cx_desc_, cy_desc_;
  hipdnnFilterDescriptor_t  w_desc_      ;
  hipdnnDropoutDescriptor_t dropout_desc_;
  hipdnnRNNDescriptor_t     rnn_desc_    ;

  Allocator* alloc_;
  int hidden_size_;
  int num_layers_;
  const int num_directions_;
  string rnn_mode_ ;
  size_t rnn_workspace_sizeInBytes_;
  void* rnn_workspace_;
  size_t rnn_trainingreserve_sizeInBytes_;
  void* rnn_trainningreserve_;
 private:
  void* dropout_workspace_;
  size_t dropout_stateSizeInBytes_;
};

template <typename T>
RNNOpCudnnBase<T>::RNNOpCudnnBase(const OpDef& def) :
    OpImpl(def),
    dropout_workspace_(NULL),
    dropout_stateSizeInBytes_(0),
    num_directions_(1),
    rnn_workspace_sizeInBytes_(0),
    rnn_trainingreserve_sizeInBytes_(0),
    rnn_workspace_(NULL), rnn_trainningreserve_(NULL) {

  checkCUDNNError(hipdnnCreateTensorDescriptor(&hx_desc_));
  checkCUDNNError(hipdnnCreateTensorDescriptor(&hy_desc_));
  checkCUDNNError(hipdnnCreateTensorDescriptor(&cx_desc_));
  checkCUDNNError(hipdnnCreateTensorDescriptor(&cy_desc_));
  checkCUDNNError(hipdnnCreateFilterDescriptor(&w_desc_));
  checkCUDNNError(hipdnnCreateDropoutDescriptor(&dropout_desc_));
  checkCUDNNError(hipdnnCreateRNNDescriptor(&rnn_desc_));
  alloc_ = GetAllocator(DeviceTypeToString(GPU));
  checkCUDNNError(hipdnnDropoutGetStatesSize(
          CudaCommon::cudnnHandle(), &dropout_stateSizeInBytes_));
  dropout_workspace_ = alloc_->Allocate<char>(dropout_stateSizeInBytes_);
  unsigned long long SEED = 1337;
  checkCUDNNError(hipdnnSetDropoutDescriptor(
        dropout_desc_,
        CudaCommon::cudnnHandle(),
        GetSingleArg<float>(def, "dropout", 1.f),
        dropout_workspace_,
        dropout_stateSizeInBytes_,
        SEED));

  hidden_size_ = GetSingleArg<int>(def, "hidden_size");
  rnn_mode_ = GetSingleArg<string>(def, "rnn_mode", "lstm");
  num_layers_ = GetSingleArg<int>(def, "num_layers", 0);
  CHECK(rnn_mode_ == "lstm") << "Currently, we only support LSTM";
  hipdnnRNNMode_t mode = HIPDNN_LSTM;
  CHECK(hidden_size_ > 0);
  checkCUDNNError(hipdnnSetRNNDescriptor(
        rnn_desc_,
        hidden_size_,
        num_layers_,
        dropout_desc_,
        HIPDNN_LSTM, //hard-coded now
        HIPDNN_LINEAR_INPUT, //hard-coded now
        HIPDNN_UNIDIRECTIONAL, //hard-coded now
        HIPDNN_LSTM, //hard-coded now
        DataTypeToCudnnType<T>::value));
}

template <typename T>
RNNOpCudnnBase<T>::~RNNOpCudnnBase() {
  /*checkCUDNNError(hipdnnDestroyTensorDescriptor(x_desc_));*/
  /*checkCUDNNError(hipdnnDestroyTensorDescriptor(y_desc_));*/
  checkCUDNNError(hipdnnDestroyTensorDescriptor(hx_desc_));
  checkCUDNNError(hipdnnDestroyTensorDescriptor(hy_desc_));
  checkCUDNNError(hipdnnDestroyTensorDescriptor(cx_desc_));
  checkCUDNNError(hipdnnDestroyTensorDescriptor(cy_desc_));
  checkCUDNNError(hipdnnDestroyFilterDescriptor(w_desc_));
  if (dropout_workspace_)
    alloc_->Deallocate<char>((char*)dropout_workspace_); 
  checkCUDNNError(hipdnnDestroyDropoutDescriptor(dropout_desc_));
  checkCUDNNError(hipdnnDestroyRNNDescriptor(rnn_desc_));
  if (!x_desc_.empty()) {
    for (auto& des : x_desc_)
      checkCUDNNError(hipdnnDestroyTensorDescriptor(des));
  }
  if (!y_desc_.empty()) {
    for (auto& des : y_desc_)
      checkCUDNNError(hipdnnDestroyTensorDescriptor(des));
  }
  if (rnn_workspace_)
    alloc_->Deallocate<char>((char*)rnn_workspace_); 
  if (rnn_trainningreserve_)
    alloc_->Deallocate<char>((char*)rnn_trainningreserve_); 
}

template <typename T>
void RNNOpCudnnBase<T>::InitCUDNN(
    int seq_length, int batch, int input_size,
    int rnn_params_count) {
  CHECK(x_desc_.empty() || x_desc_.size() == seq_length)
       << "only support fixed size corpus during iterations";
  CHECK(y_desc_.empty() || y_desc_.size() == seq_length)
       << "only support fixed size corpus during iterations";
  CHECK(seq_length > 0);
  if (x_desc_.empty()) {
    x_desc_.resize(seq_length); 
    const std::array<int, 3> dim = {batch, input_size, 1};
    const std::array<int, 3> stride = {input_size, 1, 1};
    for (int i = 0; i < seq_length; i++) {
      checkCUDNNError(hipdnnCreateTensorDescriptor(&x_desc_[i]));  
      checkCUDNNError(hipdnnSetTensorNdDescriptor(
            x_desc_[i], DataTypeToCudnnType<T>::value, 3, dim.data(), stride.data()));
    }
  }

  if (y_desc_.empty()) {
    y_desc_.resize(seq_length); 
    const std::array<int, 3> dim = {batch, hidden_size_*num_directions_, 1};
    const std::array<int, 3> stride = {hidden_size_*num_directions_, 1, 1};
    for (int i = 0; i < seq_length; i++) {
      checkCUDNNError(hipdnnCreateTensorDescriptor(&y_desc_[i]));  
      checkCUDNNError(hipdnnSetTensorNdDescriptor(
            y_desc_[i], DataTypeToCudnnType<T>::value, 3, dim.data(), stride.data()));
    }
  }
  
  {
    const std::array<int, 3> dim = {num_layers_*num_directions_, batch, hidden_size_};
    const std::array<int, 3> stride = {batch*hidden_size_, hidden_size_, 1};
    checkCUDNNError(hipdnnSetTensorNdDescriptor(
          hx_desc_, DataTypeToCudnnType<T>::value, 3, dim.data(), stride.data()));
    checkCUDNNError(hipdnnSetTensorNdDescriptor(
          hy_desc_, DataTypeToCudnnType<T>::value, 3, dim.data(), stride.data()));
    checkCUDNNError(hipdnnSetTensorNdDescriptor(
          cx_desc_, DataTypeToCudnnType<T>::value, 3, dim.data(), stride.data()));
    checkCUDNNError(hipdnnSetTensorNdDescriptor(
          cy_desc_, DataTypeToCudnnType<T>::value, 3, dim.data(), stride.data()));
  }

  {
    size_t rnn_params_sizeInBytes;
    checkCUDNNError(hipdnnGetRNNParamsSize(
          CudaCommon::cudnnHandle(),
          rnn_desc_,
          x_desc_[0],
          &rnn_params_sizeInBytes,
          DataTypeToCudnnType<T>::value));
    CHECK(rnn_params_count == rnn_params_sizeInBytes/sizeof(T));
    const std::array<int, 3> dim = {rnn_params_sizeInBytes/sizeof(T), 1, 1};
    checkCUDNNError(hipdnnSetFilterNdDescriptor(
          w_desc_,
          DataTypeToCudnnType<T>::value,
          HIPDNN_TENSOR_NCHW,
          3,
          dim.data()));
  }

  {
    size_t workspace_size;
    checkCUDNNError(hipdnnGetRNNWorkspaceSize(
          CudaCommon::cudnnHandle(),
          rnn_desc_,
          seq_length,
          x_desc_.data(),
          &workspace_size)); 
    if (workspace_size != rnn_workspace_sizeInBytes_) {
      rnn_workspace_sizeInBytes_ = workspace_size; 
      if (rnn_workspace_)
        alloc_->Deallocate<char>((char*)rnn_workspace_); 
      rnn_workspace_ = alloc_->Allocate<char>(rnn_workspace_sizeInBytes_);
    }
  }

  {
    size_t workspace_size;
    checkCUDNNError(hipdnnGetRNNTrainingReserveSize(
          CudaCommon::cudnnHandle(),
          rnn_desc_,
          seq_length,
          x_desc_.data(),
          &workspace_size)); 
    if (workspace_size != rnn_trainingreserve_sizeInBytes_) {
      rnn_trainingreserve_sizeInBytes_ = workspace_size; 
      if (rnn_trainningreserve_)
        alloc_->Deallocate<char>((char*)rnn_trainningreserve_); 
      rnn_trainningreserve_ = alloc_->Allocate<char>(rnn_trainingreserve_sizeInBytes_);
    }
  }
}

template <typename T>
class RNNOpCudnn: public RNNOpCudnnBase<T> {
 public:
  explicit RNNOpCudnn(const OpDef& def);
  void Compute(OpContext* context) override;
};

template <typename T>
RNNOpCudnn<T>::RNNOpCudnn(const OpDef& def)
  : RNNOpCudnnBase<T>(def) {}

template <typename T>
void RNNOpCudnn<T>::Compute(OpContext* context) {
  const Tensor& X = context->Input(0);
  const Tensor& W = context->Input(1);
  const Tensor& HX = context->Input(2);
  const Tensor& CX = context->Input(3);
  Tensor* Y = context->Output(0);
  Tensor* HY = context->Output(1);
  Tensor* CY = context->Output(2);

  const int seq_length = X.dims(0);
  const int batch      = X.dims(1);
  const int input_size = X.dims(2);
  const int rnn_params_count = W.count();

  this->InitCUDNN(seq_length, batch, input_size, rnn_params_count);

  checkCUDNNError(hipdnnRNNForwardTraining(
        CudaCommon::cudnnHandle(),
        seq_length,
        this->x_desc_.data(),
        X.data<T>(),
        this->hx_desc_,
        HX.data<T>(),
        this->cx_desc_,
        CX.data<T>(),
        this->w_desc_,
        W.data<T>(),
        this->y_desc_,
        Y->mutable_data<T>(),
        this->hy_desc_,
        HY->mutable_data<T>(),
        this->cy_desc_,
        CY->mutable_data<T>(),
        this->rnn_workspace_,
        this->rnn_workspace_sizeInBytes_,
        this->rnn_trainningreserve_,
        this->rnn_trainingreserve_sizeInBytes_));
}

template <typename T>
class RNNOpCudnnGrad: public RNNOpCudnnBase<T> {
 public:
  explicit RNNOpCudnnGrad(const OpDef& def);
  void Compute(OpContext* context) override;
};

template <typename T>
RNNOpCudnnGrad<T>::RNNOpCudnnGrad(const OpDef& def)
  : RNNOpCudnnBase<T>(def) {}

template <typename T>
void RNNOpCudnnGrad<T>::Compute(OpContext* context) {
  const Tensor& Y  = context->Input(0);
  const Tensor& dY = context->Input(1);
  const Tensor& X  = context->Input(2);
  const Tensor& W  = context->Input(3);
  const Tensor& HX = context->Input(4);
  const Tensor& CX = context->Input(5);

  Tensor* dX  = context->Output(0);
  Tensor* dW  = context->Output(1);
  Tensor* dHX = context->Output(2);
  Tensor* dCX = context->Output(3);

  const int seq_length = X.dims(0);
  const int batch      = X.dims(1);
  const int input_size = X.dims(2);
  const int rnn_params_count = W.count();

  this->InitCUDNN(seq_length, batch, input_size, rnn_params_count);

  checkCUDNNError(hipdnnRNNBackwardData(
        CudaCommon::cudnnHandle(),
        this->rnn_desc_,
        seq_length,
        this->y_desc_.data(),
        Y.data<T>(),
        this->y_desc_.data(),
        dY.data<T>(),
        this->hy_desc_,
        nullptr,//dhy can be nullptr, that means 0 according to cudnn manual
        this->cy_desc_,
        nullptr,//dcy can be nullptr, that means 0 according to cudnn manual
        this->w_desc_,
        W.data<T>(),
        this->hx_desc_,
        HX.data<T>(),
        this->cx_desc_,
        CX.data<T>(),
        this->x_desc_.data(),
        dX->mutable_data<T>(),
        this->hx_desc_,
        dHX->mutable_data<T>(),
        this->cx_desc_,
        dCX->mutable_data<T>(),
        this->rnn_workspace_,
        this->rnn_workspace_sizeInBytes_,
        this->rnn_trainningreserve_,
        this->rnn_trainingreserve_sizeInBytes_));
  checkCUDNNError(hipdnnRNNBackwardWeights(
        CudaCommon::cudnnHandle(),
        this->rnn_desc_,
        seq_length,
        this->x_desc_.data(),
        X.data<T>(),
        this->hx_desc_,
        HX.data<T>(),
        this->y_desc_.data(),
        Y.data<T>(),
        this->rnn_workspace_,
        this->rnn_workspace_sizeInBytes_,
        this->w_desc_,
        dW->mutable_data<T>(),
        this->rnn_trainningreserve_,
        this->rnn_trainingreserve_sizeInBytes_));
}

} //namespace backend
