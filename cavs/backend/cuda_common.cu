#include "cavs/backend/cuda_common.h"

namespace backend {

CudaCommon::CudaCommon() {
  checkCublasError(hipblasCreate(&cublasHandle_));
  checkCUDNNError(hipdnnCreate(&cudnnHandle_));
}

} //namespace backend
