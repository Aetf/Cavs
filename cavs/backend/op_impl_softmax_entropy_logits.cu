#include "hip/hip_runtime.h"
#include "cavs/backend/op_impl.h"
#include "cavs/backend/cuda_common.h"
#include "cavs/midend/devices.h"
#include "cavs/proto/tensor_shape.pb.h"
#include "cavs/util/macros_gpu.h"
#include "cavs/util/cudnn_types.h"

namespace backend {

using ::midend::Tensor;

class SoftmaxEntropyLogitsOpCudnnBase : public OpImpl {
 public:
  explicit SoftmaxEntropyLogitsOpCudnnBase(const OpDef& def);
  ~SoftmaxEntropyLogitsOpCudnnBase();

 protected:
  hipdnnTensorDescriptor_t x_desc_, y_desc_;
  hipdnnTensorDescriptor_t label_desc_;
};

SoftmaxEntropyLogitsOpCudnnBase::SoftmaxEntropyLogitsOpCudnnBase(const OpDef& def)
    : OpImpl(def) {
  checkCUDNNError(hipdnnCreateTensorDescriptor(&x_desc_));    
  checkCUDNNError(hipdnnCreateTensorDescriptor(&y_desc_));    
  checkCUDNNError(hipdnnCreateTensorDescriptor(&label_desc_));    
}

SoftmaxEntropyLogitsOpCudnnBase::~SoftmaxEntropyLogitsOpCudnnBase() {
  checkCUDNNError(hipdnnDestroyTensorDescriptor(x_desc_));
  checkCUDNNError(hipdnnDestroyTensorDescriptor(y_desc_));
  checkCUDNNError(hipdnnDestroyTensorDescriptor(label_desc_));
}

template <typename T>
class SoftmaxEntropyLogitsOpCudnn : public SoftmaxEntropyLogitsOpCudnnBase {
 public:
  explicit SoftmaxEntropyLogitsOpCudnn(const OpDef& def) 
      : SoftmaxEntropyLogitsOpCudnnBase(def) {}
  void Compute(OpContext* context) override;
};

template <typename T>
void SoftmaxEntropyLogitsOpCudnn<T>::Compute(OpContext* context) {
  const Tensor& x = context->Input(0);
  Tensor* y = context->Output(0);
  int XN = x.dims(0);
  int XC = x.dims(1);
  int XH = x.dims(2);
  int XW = x.dims(3);
  int YN = y->dims(0);
  int YC = y->dims(1);
  int YH = y->dims(2);
  int YW = y->dims(3);

  CHECK(XN == YN);
  CHECK(XC == YC);
  CHECK(XH == YH);
  CHECK(XW == YW);

  float alpha = 1.f, beta = 0.f;
  checkCUDNNError(hipdnnSetTensor4dDescriptor(x_desc_,
                  HIPDNN_TENSOR_NCHW, DataTypeToCudnnType<T>::value,
                  XN, XC, XH, XW));
  checkCUDNNError(hipdnnSetTensor4dDescriptor(y_desc_,
                  HIPDNN_TENSOR_NCHW, DataTypeToCudnnType<T>::value,
                  YN, YC, YH, YW));
  checkCUDNNError(hipdnnSoftmaxForward(CudaCommon::cudnnHandle(),
                                      HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL,
                                      &alpha, x_desc_, x.data<T>(),
                                      &beta, y_desc_, y->mutable_data<T>()));
}

template <typename T>
class SoftmaxEntropyLogitsOpCudnnGrad : public SoftmaxEntropyLogitsOpCudnnBase {
 public:
  explicit SoftmaxEntropyLogitsOpCudnnGrad(const OpDef& def) 
      : SoftmaxEntropyLogitsOpCudnnBase(def) {}
  void Compute(OpContext* context) override;
};

template <typename T>
__global__ void SoftmaxEntropyLogitsBackwardKernel(T* dx, 
    const T* y, const T* label,
    int elements, int prediction_range) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= elements)  return;
  const int label_value = static_cast<int>(label[idx/prediction_range]);
  //Through the formula of cross-entropy,
  //the derivation of dx can be denoted as follows:
  //(I deduce it for a whole noon!)
  if (label_value == idx%prediction_range)
    dx[idx] = y[idx] - 1;
  else
    dx[idx] = y[idx];
}

template <typename T>
void SoftmaxEntropyLogitsOpCudnnGrad<T>::Compute(OpContext* context) {
  const Tensor& y = context->Input(0);
  const Tensor& label = context->Input(1);
  Tensor* dx = context->Output(0);

  int NY = y.dims(0);
  int CY = y.dims(1);
  int NLabel = label.dims(0);
  int CLabel = label.dims(1);
  int NX = dx->dims(0);
  int CX = dx->dims(1);
  CHECK(NY == NX == NLabel);
  CHECK(CY == CX);
  CHECK(CLabel == 1);
  CHECK(y.dims(2) == y.dims(3) == label.dims(2) == label.dims(3)
        == dx->dims(2) == dx->dims(3) == 1);

  /*float alpha = 1.f, beta = 0.f;*/
  /*checkCUDNNError(hipdnnSetTensor4dDescriptor(x_desc_,*/
                  /*HIPDNN_TENSOR_NCHW, DataTypeToCudnnType<T>::value,*/
                  /*x.dims(0), x.dims(1), x.dims(2), x.dims(3)));*/
  /*checkCUDNNError(hipdnnSetTensor4dDescriptor(label_desc_,*/
                  /*HIPDNN_TENSOR_NCHW, DataTypeToCudnnType<T>::value,*/
                  /*label.dims(0), label.dims(1), label.dims(2), label.dims(3)));*/
  /*checkCUDNNError(hipdnnSetTensor4dDescriptor(y_desc_,*/
                  /*HIPDNN_TENSOR_NCHW, DataTypeToCudnnType<T>::value,*/
                  /*y.dims(0), y.dims(1), y.dims(2), y.dims(3)));*/
  int n = y.count();
  SoftmaxEntropyLogitsBackwardKernel<T><<<THREADS_PER_BLOCK, BLOCKS_PER_GRID(n)>>>(
        dx->mutable_data<T>(), y.data<T>(), label.data<T>(), n, CY);
  checkCudaError(hipDeviceSynchronize());
}

} //namespace backend

