#include "cavs/backend/variable_op.h"
#include "cavs/util/macros_gpu.h"

namespace backend {

template <typename T> 
struct CUDAZeroFiller {
  static void Compute(T* out, size_t n) {
    checkCudaError(hipMemset(out, 0, n*sizeof(T)));
  }
};

REGISTER_OP_BUILDER(Key("Variable").Device("GPU"), VariableOp<CUDAZeroFiller<float>, float>);

} //namespace backend
