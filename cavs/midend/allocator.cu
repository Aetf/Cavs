#include "cavs/midend/allocator.h"
/*#include "cavs/midend/devices.h"*/
#include "cavs/util/macros_gpu.h"
#include "cavs/util/op_util.h"

namespace midend {

class GPUAllocator : public Allocator {
 public:
  GPUAllocator() 
      : Allocator(DeviceTypeToString(GPU), GPU) {}    
  void* AllocateRaw(size_t nbytes) override {
    VLOG(V_DEBUG) << "allocating " << nbytes << " bytes";
    void* ptr = NULL;
    checkCudaError(hipMalloc(&ptr, nbytes)); 
    checkCudaError(hipMemset(ptr, 0, nbytes)); 
    CHECK_NOTNULL(ptr);
    return ptr;
  }
  void DeallocateRaw(void* buf) override {
    checkCudaError(hipFree(buf));
  }
  void InitWithZero(void* buf, size_t nbytes) override {
    checkCudaError(hipMemsetAsync(buf, 0, nbytes, hipStreamDefault));
  }
};

Allocator* gpu_allocator() {
  static GPUAllocator gpu_alloc;
  return &gpu_alloc;
}

REGISTER_STATIC_ALLOCATOR("GPU", gpu_allocator());

} //namespace midend
