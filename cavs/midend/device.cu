#include "cavs/midend/devices.h"
#include "cavs/midend/macros_gpu.h"

namespace cavs {

void DeviceContext::MemcpyHostToDevice(Tensor* out, const Tensor& inp) {
  checkCudaError(hipMemcpy(out->buf_->data(), inp.buf_->data(), 
                 inp.buf_->size(), 
                 hipMemcpyHostToDevice));
}

void DeviceContext::MemcpyDeviceToHost(Tensor* out, const Tensor& inp) {
  checkCudaError(hipMemcpy(out->buf_->data(), inp.buf_->data(), 
                 inp.buf_->size(), 
                 hipMemcpyDeviceToHost));

}

} //namespace cavs
