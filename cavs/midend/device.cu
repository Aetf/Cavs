#include "cavs/midend/devices.h"
#include "cavs/util/macros_gpu.h"

namespace midend {

void DeviceContext::MemcpyHostToDevice(Tensor* out, const Tensor& inp) {
  checkCudaError(hipMemcpy(out->buf_->data(), inp.buf_->data(), 
                 inp.buf_->size(), 
                 hipMemcpyHostToDevice));
}

void DeviceContext::MemcpyDeviceToHost(Tensor* out, const Tensor& inp) {
  checkCudaError(hipMemcpy(out->buf_->data(), inp.buf_->data(), 
                 inp.buf_->size(), 
                 hipMemcpyDeviceToHost));

}

} //namespace midend 
