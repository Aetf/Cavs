#include "cavs/core/allocator.h"
#include "cavs/core/macros_gpu.h"

namespace cavs {

class GPUAllocator : public Allocator {
 public:
  GPUAllocator() : Allocator() {}    
  string Name() override { return "GPU"; }

  void* AllocateRaw(size_t nbytes) override {
    void* ptr = NULL;
    checkCudaError(hipMalloc(&ptr, nbytes)); 
    return ptr;
  }

  void DeallocateRaw(void* buf) override {
    checkCudaError(hipFree(buf));
  }
};

Allocator* gpu_allocator() {
  static GPUAllocator gpu_alloc;
  return &gpu_alloc;
}
REGISTER_STATIC_ALLOCATOR("GPU", gpu_allocator());

}
