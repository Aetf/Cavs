#include "functions.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <glog/logging.h>
#include <gflags/gflags.h>

#define checkCublasError(status)                             \
        do {                                                 \
            if (status != HIPBLAS_STATUS_SUCCESS) {           \
                LOG(FATAL) << "CUDA failure: "               \
                           << status;                        \
            }                                                \
        }while(0)
#define checkCudaError(status)                               \
        do {                                                 \
            if (status != hipSuccess) {                     \
                LOG(FATAL) << "CUDA failure: "               \
                           << hipGetErrorString(status);    \
            }                                                \
       }while(0)

class Common{
public:
    inline static hipblasHandle_t cublasHandle(){return Get()->cublasHandle_;}
private:
    Common();
    static Common* Get() {static Common c; return &c;}
    hipblasHandle_t cublasHandle_;
};

Common::Common(){
    checkCublasError(hipblasCreate(&cublasHandle_));
}

template<>
Tensor<float>::Tensor(int c){
    capacity_ = c*sizeof(float);
    cpu_buf_ = malloc(capacity_);
    checkCudaError(hipMalloc(&gpu_buf_, capacity_));
}

template<>
Tensor<double>::Tensor(int c){
    capacity_ = c*sizeof(double);
    cpu_buf_ = malloc(capacity_);
    checkCudaError(hipMalloc(&gpu_buf_, capacity_));
}

template<>
void Tensor<float>::sync2d(){
    checkCudaError(hipMemcpy(gpu_buf_, cpu_buf_, capacity_, hipMemcpyHostToDevice));
}

template<>
void Tensor<double>::sync2d(){
    checkCudaError(hipMemcpy(gpu_buf_, cpu_buf_, capacity_, hipMemcpyHostToDevice));
}

template<>
void Tensor<float>::sync2h(){
    checkCudaError(hipMemcpy(cpu_buf_, gpu_buf_, capacity_, hipMemcpyDeviceToHost));
}

template<>
void Tensor<double>::sync2h(){
    checkCudaError(hipMemcpy(cpu_buf_, gpu_buf_, capacity_, hipMemcpyDeviceToHost));
}

template <>
void cublasWrapper<float>(const bool TransA, const bool TransB, 
        const int M, const int N, const int K, 
        const float alpha, const float *A, const float *B,
        const float beta, float *C){
    int lda = (TransA == false) ? K : M;
    int ldb = (TransB == false) ? N : K;
    hipblasOperation_t cuTransA =
        (TransA == false) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    hipblasOperation_t cuTransB =
        (TransB == false) ? HIPBLAS_OP_N : HIPBLAS_OP_T;

    checkCublasError(hipblasSgemm(Common::cublasHandle(), cuTransB, cuTransA,
        N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void cublasWrapper<double>(const bool TransA, const bool TransB, 
        const int M, const int N, const int K, 
        const double alpha, const double *A, const double *B,
        const double beta, double *C){
    int lda = (TransA == false) ? K : M;
    int ldb = (TransB == false) ? N : K;
    hipblasOperation_t cuTransA =
        (TransA == false) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    hipblasOperation_t cuTransB =
        (TransB == false) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    checkCublasError(hipblasDgemm(Common::cublasHandle(), cuTransB, cuTransA,
        N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

